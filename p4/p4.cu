#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/types.h>
#include <sys/time.h>
#define VERBOSE 1
__global__ void checkMatchOnDevice(char *fileBuffer, char* searchString, int* matchArray, int numBytes,size_t searchSize,int* matchStartArray, int* matchEndArray)
{
  extern __shared__ int sdata[];
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int rangeStart;
  int rangeEnd; 
  rangeStart = idx*numBytes;
  rangeEnd = rangeStart + numBytes;
  int i,j;
  int numMatches;
  int foundMatch;
  int firstMatch = 1;
  int firstMatchIndex = -1;
  int lastMatchIndex = -1;
  unsigned int tid = threadIdx.x; 

  for(numMatches=0, i = rangeStart; i < rangeEnd; i++)
  {
    foundMatch = 1;
    for(j = 0; j < searchSize; j++)
    {
      int index = i+j;
      if(fileBuffer[index] != searchString[j])
      {
        foundMatch = 0;
        break;
      }
    }
    if(foundMatch)
    {
      numMatches++;
      if(firstMatch)
      {
	firstMatchIndex = i;
        firstMatch = 0;
      }
      lastMatchIndex = i+searchSize;
      i+=searchSize-1;
    }
  }
  matchStartArray[idx] = firstMatchIndex;
  matchEndArray[idx] = lastMatchIndex;
  //matchArray[idx] = numMatches;
  sdata[tid] = numMatches;

  __syncthreads();
  for(unsigned int s=1;s<blockDim.x;s *= 2){
    int index = s*2*tid;
    if((index+s) <  blockDim.x){
	    sdata[index] += sdata[index + s];
    }
    __syncthreads();
  }
  if(tid==0) matchArray[blockIdx.x]=sdata[0];  
}
__global__ void cumulateOnDevice(int* matchArray, int noOfThreads, int* outArray)
{
  extern __shared__ int sdata2[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int lim;// = (noOfThreads > blockDim.x) ? blockDim.x:noOfThreads;
  if(i<noOfThreads){
	sdata2[tid] = matchArray[i];
	__syncthreads();

	for(unsigned int s=1;s<blockDim.x;s*=2){
	  int index = s*2*tid;
	
	//__syncthreads();
	  if((index+s) < noOfThreads){
	    sdata2[index] += sdata2[index+s];
	  }
	 __syncthreads();
	}
	if(tid == 0) matchArray[blockIdx.x] = sdata2[0];
  }
}
int main(int argc, char *argv[])
{
  struct timeval cpuStart,cpuEnd;
  char* searchString = (char*)malloc(sizeof(char*)*80);
  char* fileBuffer = (char*)malloc(sizeof(char*)*10000000);
  int nBlocks;
  int threadsPerBlock;
  if(argc != 4)
  {
    printf("Usage: stringSearch \"Key\" numBlocks threadsPerBlock < inputFile\n");
    exit(0);
  }
  else
  {
    searchString = argv[1];
    nBlocks = atoi(argv[2]);
    threadsPerBlock = atoi(argv[3]);
    #ifdef VERBOSE
    printf("Search String: %s\n",searchString);
    #endif
    int ptr;
    for(ptr = 0; !feof(stdin);)
    {
      ptr+= fread(&(fileBuffer[ptr]),1,1,stdin);
    }
    char *deviceFileBuffer;                 // pointer to device memory
    char *deviceSearchBuffer;                 // pointer to device memory
    int *matchArray;
    int *outArray;
    int *hostMatchArray;
    int *hostMatchStartArray;
    int *hostMatchEndArray;
    int *matchStartArray;
    int *matchEndArray;
    int fileSize = ptr;
    //printf("FileSize: %d",strlen(fileBuffer));
    size_t searchSize = strlen(searchString);
    hipEvent_t start, stop; 
    float time; 
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    gettimeofday(&cpuStart, NULL);
    // allocate file buffer space on device 
    hipMalloc((void **) &deviceFileBuffer, fileSize);
    // allocate search string space on device 
    hipMalloc((void **) &deviceSearchBuffer, searchSize);
    // copy data from host to device
    hipMemcpy(deviceFileBuffer, fileBuffer, fileSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceSearchBuffer, searchString, searchSize, hipMemcpyHostToDevice);
    // do calculation on device:
    // Part 1 of 2. Compute execution configuration
    size_t numThreads = nBlocks*threadsPerBlock;
    int numBytesPerThread = fileSize/numThreads; 
    //Allocate match array space on device
    hipMalloc((void **) &matchArray, sizeof(int)*numThreads);
    hipMalloc((void **) &outArray, sizeof(int)*numThreads);
    hostMatchArray = (int*)malloc(sizeof(int)*numThreads);
    hipMalloc((void **) &matchStartArray, sizeof(int)*numThreads);
    hostMatchStartArray = (int*)malloc(sizeof(int)*numThreads);
    hipMalloc((void **) &matchEndArray, sizeof(int)*numThreads);
    hostMatchEndArray = (int*)malloc(sizeof(int)*numThreads);
    //Init array to 0
    int i;
    for(i = 0; i < numThreads; i++)
    {
      hostMatchArray[i] = 0; 
      hostMatchStartArray[i] = -1; 
      hostMatchEndArray[i] = -1; 
    }
    hipMemcpy(matchArray, hostMatchArray, numThreads, hipMemcpyHostToDevice);
    hipMemcpy(matchStartArray, hostMatchStartArray, numThreads, hipMemcpyHostToDevice);
    hipMemcpy(matchEndArray, hostMatchEndArray, numThreads, hipMemcpyHostToDevice);
    //printf("Number of threads:%d, Number of blocks:%d, Num Threads Per Block:%d, Num Bytes Per Thread:%d\n",numThreads,nBlocks,threadsPerBlock,numBytesPerThread);
    // Part 2 of 2. Call call checkMatchOnDevice kernel
    hipEventRecord( start, 0 );
    checkMatchOnDevice <<< nBlocks, threadsPerBlock >>> (deviceFileBuffer, deviceSearchBuffer, matchArray,numBytesPerThread,searchSize,matchStartArray,matchEndArray);
    int newNBlocks = (nBlocks/threadsPerBlock)+1;printf("\nNew Blocks:%d",nBlocks);
    hipDeviceSynchronize();
    cumulateOnDevice <<< newNBlocks, threadsPerBlock ,threadsPerBlock * sizeof(int)>>> (matchArray,nBlocks,outArray);
    //hipMemcpy(matchArray, outArray, sizeof(int)*numThreads, hipMemcpyDeviceToDevice);
    hipEventRecord( stop, 0 ); 
    hipEventSynchronize( stop ); 
    hipEventElapsedTime( &time, start, stop ); 
    hipEventDestroy( start ); 
    hipEventDestroy( stop ); 
    hipDeviceSynchronize();
    // Retrieve result from device and store in host array
    hipMemcpy(hostMatchArray, matchArray, sizeof(int)*numThreads, hipMemcpyDeviceToHost);
    hipMemcpy(hostMatchStartArray, matchStartArray, sizeof(int)*numThreads, hipMemcpyDeviceToHost);
    hipMemcpy(hostMatchEndArray, matchEndArray, sizeof(int)*numThreads, hipMemcpyDeviceToHost);
    int total = 0;
    for(i = 0; i < numThreads; i++)
    {
	total += hostMatchArray[i];
        printf("%d)%d\n",i,hostMatchArray[i]);
    }
	total = hostMatchArray [0];
    //Overlap check, commented out for hw2  
/*  for(i = 0; i < numThreads; i++)
    {
	if(hostMatchEndArray[i] != -1 && hostMatchStartArray[i+1] != -1)
	{
		if(hostMatchEndArray[i] - hostMatchStartArray[i+1] < 0)
			total--;	
	}
	//printf("%d)%d\n",i,hostMatchStartArray[i]);
	//printf("start:%d,end:%d\n",hostMatchStartArray[i],hostMatchEndArray[i]);
    }*/
    
    gettimeofday(&cpuEnd, NULL);
   // float totalTime = (cpuEnd - cpuStart);
    //printf("Number of threads:%d, Number of blocks:%d, Num Threads Per Block:%d, Num Bytes Per Thread:%d\n",numThreads,nBlocks,threadsPerBlock,numBytesPerThread);
    //printf("numOfThread: %4d matchCount: %4d CPUrunningTime: %8ld\n", blocksize, thread_num, sum, q.tv_usec - p.tv_usec + (q.tv_sec-p.tv_sec)*1000000);
    #ifdef VERBOSE
    printf("Completed Successfully! Number of blocks:%d Number of threads per block:%d Num Threads: %d Matches:%d CPU Time:%8ld GPU Time:%f\n\n",nBlocks,threadsPerBlock,numThreads,total,cpuEnd.tv_usec - cpuStart.tv_usec + (cpuEnd.tv_sec - cpuStart.tv_sec ),time);
    #else
    printf("%d %f\n\n",numThreads,time);
    //printf("%d %8ld\n\n",numThreads,cpuEnd.tv_usec - cpuStart.tv_usec + (cpuEnd.tv_usec - cpuStart.tv_usec) );
    #endif
    hipFree(matchArray);
    hipFree(outArray);
  }
  
}
